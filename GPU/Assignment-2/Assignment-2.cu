#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>

const int N = 100000;

__global__ void dotProduct(float* a, float* b, float* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    float tempResult = 0.0f;
    if (idx < N) {
        tempResult += a[idx] * b[idx];
    }

    atomicAdd(result, tempResult);
}

int main() {
    float* h_a, * h_b, * d_a, * d_b, * d_result;
    float result = 0.0f;

    h_a = new float[N];
    h_b = new float[N];

    for (int i = 0; i < N; ++i) {
        h_a[i] = 1;
        h_b[i] = 1;
    }

    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    // Create CUDA events for measuring time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    dotProduct << <numBlocks, numThreadsPerBlock >> > (d_a, d_b, d_result);

    // Record the stop event
    hipEventRecord(stop);

    // Synchronize to make sure the stop event finishes
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result back from the device to the host
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Output results
    std::cout << "Dot Product: " << result << std::endl;
    std::cout << "Kernel execution time: " << milliseconds << " ms\n";

    // Clean Memory
    delete[] h_a;
    delete[] h_b;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}