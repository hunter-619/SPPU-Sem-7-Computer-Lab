#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

const int N = 512;

__global__ void matrixTranspose(float* A, float* B) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    B[col * N + row] = A[row * N + col];
}

__global__ void matrixMultiply(float* A, float* B, float* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0f;

    for (int k = 0; k < N; ++k) {
        sum += A[row * N + k] * B[k * N + col];
    }

    C[row * N + col] = sum;
}

int main() {
    float* h_A, * h_B, * h_C;
    float* d_A, * d_B, * d_C;

    h_A = new float[N * N];
    h_B = new float[N * N];
    h_C = new float[N * N];

    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(N / 16, N / 16);

    matrixMultiply << <gridSize, blockSize >> > (d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}